#include "hip/hip_runtime.h"
/* Matrix normalization.
* Compile with "nvcc matrixNormCuda.c -lm"
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

/* Program Parameters */
#define N 6000  /* Matrix size */

/* Matrices */
volatile float A[N][N], B[N][N];

/* CUDA arrays */
volatile float *A_d, *B_d;


/* Initialize A and B*/
void initialize_inputs() {
    int row, col;
    
    srand((unsigned)time(NULL));
    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            A[row][col] = (float)rand() / 32768.0;
            B[row][col] = 0.0;
        }
    }
    
}


/* Kernel function */

__global__ void matrixNorm(float mu, float sigma, int N) {
    for (row=0; row < N; row++) {
        if (sigma == 0.0)
            B_d[row*N + col] = 0.0;
        else
            B_d[row*N + col] = (A_d[row*N + col] - mu) / sigma;
    }
}



int main(int argc, char **argv) {
    /* Timing variables */
    struct timeval start, stop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    unsigned long long runtime;
    int col, row;
    float mu, sigma;
    
    /* Initialize A and B */
    initialize_inputs();
    
    
    /* Start Clock */
    printf("\n---------------------------------------------\n");
    printf("Matrix size N = %d", N);
    printf("\nStarting clock.\n\n");
    gettimeofday(&start, &tzdummy);
    
    printf("Computing Serially.\n");
    
    /*allocating GPU space*/
    hipError_t err1 = hipMalloc((void **) &A_d, N);
    hipError_t err2 = hipMalloc((void **) &B_d, N);

    /*transfer data from host to device*/
    hipMemcpy(A_d,A,N*N,hipMemcpyHostToDevice);

    /* Kernal Matrix Normalization */
    for (col=0; col < N; col++) {
        mu = 0.0;
        for (row=0; row < N; row++)
            mu += A[row][col];
        mu /= (float) N;
        sigma = 0.0;
        for (row=0; row < N; row++)
            sigma += powf(A[row][col] - mu, 2.0);
        sigma /= (float) N;
        sigma = sqrt(sigma);
        matrixNorm<<< >>>(mu,sigma, N);
    }

    /*transfer data from device to host*/
    hipMemcpy(B_d,B,N*N,hipMemcpyDeviceToHost);
    
    /*deallocating GPU space*/
    hipFree(A_d);
    hipFree(B_d);
    
    /* Stop Clock */
    gettimeofday(&stop, &tzdummy);
    runtime = (unsigned long long)(stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec);
    
    
    /* Display timing results */
    printf("Runtime = %g ms.\n", (float)runtime/(float)1000);
    printf("\nStopped clock.");
    printf("\n---------------------------------------------\n");
    
    exit(0);
}

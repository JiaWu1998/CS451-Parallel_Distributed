#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 5
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    printf("hi");
    for(int i = 0; i < n; i ++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    for(int i = 0; i < N; i++){
        printf("%f,",b[i]);
    }
    printf("\n");

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        printf("%f,",out[i]);
    }
    printf("\n");

    // Verification
    // for(int i = 0; i < N; i++){
    //     assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    // }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
